#include "hip/hip_runtime.h"

/* 
Use CUDA functions to calculate block size 
*/ 

#include <iostream>
#include <stdlib.h>
#include <cmath> 
#include <string>
#include <ctime> 
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// Position struct contains x and y coordinates 
struct Position {
    float x, y; 

    std::string toString() {
        return "(" + std::to_string(x) + "," + std::to_string(y) + ")"; 
    }

    __device__ __host__ void operator+=(const Position& a) {
        x = x + a.x;
        y = y + a.y; 
    }

    __device__ __host__ void operator=(const Position& a) {
        x = a.x; 
        y = a.y; 
    }
}; 

// Particle struct has current location, best location and velocity 
struct Particle {
    Position best_position; 
    Position current_position; 
    Position velocity; 
    float best_value; 
};


const unsigned int N = 5000; 
const unsigned int ITERATIONS = 1000; 
const float SEARCH_MIN = -1000.0f; 
const float SEARCH_MAX = 1000.0f; 
const float w = 0.9f; 
const float c_ind = 1.0f; 
const float c_team = 2.0f; 

// return a random float between low and high 
float randomFloat(float low, float high) {
    float range = high-low; 
    float pct = static_cast <float>(rand()) / static_cast <float>(RAND_MAX); 
    return low + pct * range; 
}

// function to optimize 
__device__ __host__ float calcValue(Position p) {
    return pow(p.x, 2) + pow(p.y, 2); 
}

// Initialize state for random numbers 
__global__ void init_kernel(hiprandState *state, long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    hiprand_init(seed, idx, 0, state);
}

// Returns the index of the particle with the best position
__global__ void updateTeamBestIndex(Particle *d_particles, float *team_best_value, int *team_best_index, int N) {
    __shared__ float best_value; 
    __shared__ int best_index; 
    best_value = d_particles[0].best_value;
    best_index = 0; 
    __syncthreads(); 
    
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    if (idx < N) {
        if (d_particles[idx].best_value < best_value) {
            best_value = d_particles[idx].best_value; 
            best_index = idx; 
            __syncthreads(); 
        }
    }
    *team_best_value = best_value; 
    *team_best_index = best_index; 
}


// Update velocity for all particles 
__global__ void updateVelocity(Particle* d_particles, int *team_best_index, float w, float c_ind, float c_team, int N, hiprandState *state) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 

    __shared__ float best_x, best_y; 
    best_x = d_particles[*team_best_index].best_position.x; 
    best_y = d_particles[*team_best_index].best_position.y; 
    __syncthreads(); 

    if (idx < N) {
        float r_ind = hiprand_uniform(state);
        float r_team = hiprand_uniform(state);
        d_particles[idx].velocity.x = w * d_particles[idx].velocity.x + 
                       r_ind * c_ind * (d_particles[idx].best_position.x - d_particles[idx].current_position.x) + 
                       r_team * c_team * (best_x - d_particles[idx].current_position.x); 
        d_particles[idx].velocity.y = w * d_particles[idx].velocity.y + 
                       r_ind * c_ind * (d_particles[idx].best_position.y - d_particles[idx].current_position.y) + 
                       r_team * c_team * (best_y - d_particles[idx].current_position.y); 
    }
}

__global__ void updatePosition(Particle *d_particles, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    if (idx < N) {
        d_particles[idx].current_position += d_particles[idx].velocity; 
        float newValue = calcValue(d_particles[idx].current_position); 
        if (newValue < d_particles[idx].best_value) {
            d_particles[idx].best_value = newValue; 
            d_particles[idx].best_position = d_particles[idx].current_position; 
        }
    }
}


int main(void) {
    // for timing 
    long start = std::clock();

    // Random seed for cpu 
    std::srand(std::time(NULL)); 
    // Random seed for gpu 
    hiprandState *state; 
    hipMalloc(&state, sizeof(hiprandState)); 
    init_kernel<<<1,1>>>(state, clock()); 

    // Initialize particles 
    Particle *particles; 
    size_t particleSize = sizeof(Particle) * N; 

    // initialize variables for team best 
    int *team_best_index; 
    float *team_best_value; 

    // Allocate particles in unified memory 
    hipMallocManaged(&particles, particleSize);
    hipMallocManaged(&team_best_index, sizeof(int)); 
    
    // Allocate team_best_value for gpu only 
    hipMalloc(&team_best_value, sizeof(float)); 

    // Prefetch data to the GPU 
    int device = hipGetDevice(&device); 
    hipMemPrefetchAsync(team_best_index, sizeof(int), device, NULL); // ptr, size_t, device, stream 

    // Memory hints 
    hipMemAdvise(particles, particleSize, hipMemAdviseSetPreferredLocation, hipCpuDeviceId); // start on cpu

    //  Initialize particles on host 
    for (int i = 0; i < N; i++) {
        // Random starting position
        particles[i].current_position.x = randomFloat(SEARCH_MIN, SEARCH_MAX); 
        particles[i].current_position.y = randomFloat(SEARCH_MIN, SEARCH_MAX); 
        particles[i].best_position.x = particles[i].current_position.x; 
        particles[i].best_position.y = particles[i].current_position.y; 
        particles[i].best_value = calcValue(particles[i].best_position); 
        // Random starting velocity 
        particles[i].velocity.x = randomFloat(SEARCH_MIN, SEARCH_MAX); 
        particles[i].velocity.y = randomFloat(SEARCH_MIN, SEARCH_MAX); 
    }

    // Prefetch particles to gpu 
    hipMemPrefetchAsync(particles, particleSize, device, NULL); 

    // Initialize team best index and value 
    updateTeamBestIndex<<<1,1>>>(particles, team_best_value, team_best_index, N); 

    // assign thread and blockcount 
    int blockSize = 32; 
    int gridSize = (N + blockSize - 1) / blockSize; 

    // For i in interations 
    for (int i = 0; i < ITERATIONS; i++) {
        updateVelocity<<<gridSize, blockSize>>>(particles, team_best_index, w, c_ind, c_team, N, state); 
        updatePosition<<<gridSize, blockSize>>>(particles, N); 
        updateTeamBestIndex<<<gridSize, blockSize>>>(particles, team_best_value, team_best_index, N); 
    }

    // Wait for gpu to finish computation 
    hipDeviceSynchronize(); 

    // Prefetch particles and best index back 
    hipMemPrefetchAsync(particles, particleSize, hipCpuDeviceId); 
    hipMemPrefetchAsync(team_best_index, sizeof(int), hipCpuDeviceId); 

    // Stop clock
    long stop = std::clock(); 
    long elapsed = (stop - start) * 1000 / CLOCKS_PER_SEC;

    // print results 
    std::cout << "Ending Best: " << std::endl;
    std::cout << "Team best value: " << particles[*team_best_index].best_value << std::endl;
    std::cout << "Team best position: " << particles[*team_best_index].best_position.toString() << std::endl; 
    
    std::cout << "Run time: " << elapsed << "ms" << std::endl;

    hipFree(particles); 
    hipFree(team_best_index); 
    hipFree(team_best_value); 
    hipFree(state);
    return 0; 
}