
/* 
Use CUDA functions to calculate block size 
*/ 

#include <iostream>
#include <stdlib.h>
#include <cmath> 
#include <string>
#include <ctime> 
#include <hip/hip_runtime.h> 

#include <hiprand/hiprand_kernel.h>

// Position struct contains x and y coordinates 
struct Position {
    float x, y; 

    std::string toString() {
        return "(" + std::to_string(x) + "," + std::to_string(y) + ")"; 
    }

    __device__ __host__ void operator+=(const Position& a) {
        x = x + a.x;
        y = y + a.y; 
    }

    __device__ __host__ void operator=(const Position& a) {
        x = a.x; 
        y = a.y; 
    }
}; 

// Particle struct has current location, best location and velocity 
struct Particle {
    Position best_position; 
    Position current_position; 
    Position velocity; 
    float best_value; 
};


const unsigned int N = 5000; 
const unsigned int ITERATIONS = 1000; 
const float SEARCH_MIN = -1000.0f; 
const float SEARCH_MAX = 1000.0f; 
const float w = 0.9f; 
const float c_ind = 1.0f; 
const float c_team = 2.0f; 

// return a random float between low and high 
float randomFloat(float low, float high) {
    float range = high-low; 
    float pct = static_cast <float>(rand()) / static_cast <float>(RAND_MAX); 
    return low + pct * range; 
}

// function to optimize 
__device__ __host__ float calcValue(Position p) {
    return pow(p.x, 2) + pow(p.y, 2); 
}

// Initialize state for random numbers 
__global__ void init_kernel(hiprandState *state, long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    hiprand_init(seed, idx, 0, state);
}

// Returns the index of the particle with the best position
__global__ void updateTeamBestIndex(Particle *d_particles, float *d_team_best_value, int *d_team_best_index, int N) {
    __shared__ float best_value; 
    __shared__ int best_index; 
    best_value = d_particles[0].best_value;
    best_index = 0; 
    __syncthreads(); 
    
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    if (idx < N) {
        if (d_particles[idx].best_value < best_value) {
            best_value = d_particles[idx].best_value; 
            best_index = idx; 
            __syncthreads(); 
        }
    }
    *d_team_best_value = best_value; 
    *d_team_best_index = best_index; 
}


// Update velocity for all particles 
__global__ void updateVelocity(Particle* d_particles, int *d_team_best_index, float w, float c_ind, float c_team, int N, hiprandState *state) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 

    __shared__ float best_x, best_y; 
    best_x = d_particles[*d_team_best_index].best_position.x; 
    best_y = d_particles[*d_team_best_index].best_position.y; 
    __syncthreads(); 

    if (idx < N) {
        float r_ind = hiprand_uniform(state);
        float r_team = hiprand_uniform(state);
        d_particles[idx].velocity.x = w * d_particles[idx].velocity.x + 
                       r_ind * c_ind * (d_particles[idx].best_position.x - d_particles[idx].current_position.x) + 
                       r_team * c_team * (best_x - d_particles[idx].current_position.x); 
        d_particles[idx].velocity.y = w * d_particles[idx].velocity.y + 
                       r_ind * c_ind * (d_particles[idx].best_position.y - d_particles[idx].current_position.y) + 
                       r_team * c_team * (best_y - d_particles[idx].current_position.y); 
    }
}

__global__ void updatePosition(Particle *d_particles, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    if (idx < N) {
        d_particles[idx].current_position += d_particles[idx].velocity; 
        float newValue = calcValue(d_particles[idx].current_position); 
        if (newValue < d_particles[idx].best_value) {
            d_particles[idx].best_value = newValue; 
            d_particles[idx].best_position = d_particles[idx].current_position; 
        }
    }
}


int main(void) {
    // Random seed for cpu 
    std::srand(std::time(NULL)); 
    // Random seed for gpu 
    hiprandState *state; 
    hipMalloc(&state, sizeof(hiprandState)); 
    init_kernel<<<1,1>>>(state, clock()); 

    // Initialize particles 
    Particle* h_particles = new Particle[N]; 
    Particle* d_particles;  // for the gpu 

    for (int i = 0; i < N; i++) {
        // Random starting position
        h_particles[i].current_position.x = randomFloat(SEARCH_MIN, SEARCH_MAX); 
        h_particles[i].current_position.y = randomFloat(SEARCH_MIN, SEARCH_MAX); 
        h_particles[i].best_position.x = h_particles[i].current_position.x; 
        h_particles[i].best_position.y = h_particles[i].current_position.y; 
        h_particles[i].best_value = calcValue(h_particles[i].best_position); 
        // Random starting velocity 
        h_particles[i].velocity.x = randomFloat(SEARCH_MIN, SEARCH_MAX); 
        h_particles[i].velocity.y = randomFloat(SEARCH_MIN, SEARCH_MAX); 
    }

    // Allocate memory + copy data to gpu 
    size_t particleSize = sizeof(Particle) * N; 
    hipMalloc((void **)&d_particles, particleSize); 
    hipMemcpy(d_particles, h_particles, particleSize, hipMemcpyHostToDevice); // dest, source, size, direction

    // initialize variables for gpu 
    int *d_team_best_index; 
    float *d_team_best_value; 

    // Allocate gpu memory 
    hipMalloc((void **)&d_team_best_index, sizeof(int)); 
    hipMalloc((void **)&d_team_best_value, sizeof(float)); 

    // Initialize team best index and value 
    updateTeamBestIndex<<<1,1>>>(d_particles, d_team_best_value, d_team_best_index, N); 

    // assign thread and blockcount 
    int blockSize = 32; 
    int gridSize = (N + blockSize - 1) / blockSize; 

    // for timing 
    long start = std::clock();
    // For i in interations 
    for (int i = 0; i < ITERATIONS; i++) {
        updateVelocity<<<gridSize, blockSize>>>(d_particles, d_team_best_index, w, c_ind, c_team, N, state); 
        updatePosition<<<gridSize, blockSize>>>(d_particles, N); 
        updateTeamBestIndex<<<gridSize, blockSize>>>(d_particles, d_team_best_value, d_team_best_index, N); 
    }

    long stop = std::clock(); 
    long elapsed = (stop - start) * 1000 / CLOCKS_PER_SEC;

    // copy best particle back to host 
    int team_best_index; 
    hipMemcpy(&team_best_index, d_team_best_index, sizeof(int), hipMemcpyDeviceToHost); 
    
    // copy particle data back to host 
    hipMemcpy(h_particles, d_particles, particleSize, hipMemcpyDeviceToHost);

    // print results 
    std::cout << "Ending Best: " << std::endl;
    std::cout << "Team best value: " << h_particles[team_best_index].best_value << std::endl;
    std::cout << "Team best position: " << h_particles[team_best_index].best_position.toString() << std::endl; 
    
    std::cout << "Run time: " << elapsed << "ms" << std::endl;

    hipFree(d_particles); 
    hipFree(d_team_best_index); 
    hipFree(d_team_best_value); 
    hipFree(state); 
    return 0; 
}